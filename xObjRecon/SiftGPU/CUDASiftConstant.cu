#include "SiftCameraParams.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

//__constant__ SiftCameraParams c_siftCameraParams;
__device__ SiftCameraParams *c_siftCameraParams;
//__device__ SiftCameraParams d_siftCameraParams;

extern "C" void updateConstantSiftCameraParams(const SiftCameraParams& params) {
	
	size_t size = sizeof(SiftCameraParams);
	//checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(c_siftCameraParams)));
	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_siftCameraParams), &params, size, 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc(&c_siftCameraParams, size));
	checkCudaErrors(hipMemcpy(c_siftCameraParams, &params, size, hipMemcpyHostToDevice));
	//SiftCameraParams c_siftCameraParams_host;
	//memset(&c_siftCameraParams_host, 0, size);
	//hipMemcpyFromSymbol(&c_siftCameraParams_host, HIP_SYMBOL(c_siftCameraParams), size, 0, hipMemcpyDeviceToHost);
	//std::cout << c_siftCameraParams_host.m_depthWidth << std::endl;
	
#ifdef DEBUG
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckMsg(__FUNCTION__);
#endif

}