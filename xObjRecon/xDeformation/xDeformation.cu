#include "hip/hip_runtime.h"
#include "stdafx.h"

#include "xDeformation.h"

#include "xDeformation/Cuda/xDeformationCudaFuncs.cuh"
#include "Helpers/xUtils.h"
#include "Helpers/UtilsMath.h"
#include "Helpers/InnorealTimer.hpp"
#include "Helpers/xGlobalStats.h"
#include "xMeshEdgeSample/xPointCloudEdgeSample.hpp"
#include "SiftGPU/xSift.h"
#include "GMS/xGMS.h"

#include <cassert>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>
#include <windows.h>
//#include <Eigen/Eigen>
#include "GNSolver.h"
#include "InputData.h"

xDeformation::xDeformation(int& fragIdx, VBOType* dVboCuda)
	: m_dVboCuda(dVboCuda),
	  m_fragIdx(fragIdx),
	  m_width(Resolution::getInstance().width()),
	  m_height(Resolution::getInstance().height()),
	  m_loopClosureNum(0),
	  m_matchingPointNum(0),
	  m_matchingPointsNumDescriptor(0),
	  m_matchingPointsNumNearest(0)
{
	srand((unsigned)time(NULL));

	allocEssentialCUDAMemory();

	m_isFragValid = std::vector<int>(MAX_FRAG_NUM, 3);

	//m_grayImgDevice = cv::cuda::GpuMat(m_height, m_width, CV_8UC1);
	m_dVerticalBlurImg = cv::cuda::GpuMat(m_height, m_width, CV_32FC1);
	m_dHorizontalBlurImg = cv::cuda::GpuMat(m_height, m_width, CV_32FC1);

	cv::getDerivKernels(m_kxRow, m_kyRow, 1, 0, CV_SCHARR, true);
	cv::getDerivKernels(m_kxCol, m_kyCol, 0, 1, CV_SCHARR, true);
#if 0
	std::cout << m_kx_row << std::endl;
	std::cout << m_ky_row << std::endl;
	std::cout << m_kx_col << std::endl;
	std::cout << m_ky_col << std::endl;
#endif

	m_inputData = new InputData();
	//const SolverPara param(true, 10.0f, false, 1.0f, false, 60.0f, false, 1000.0f);
	const SolverPara param(GlobalState::getInstance().m_withGeoTerm, pow(GlobalState::getInstance().m_weightGeo, 2),
												 GlobalState::getInstance().m_withPhotoTerm, pow(GlobalState::getInstance().m_weightPhoto, 2),
												 GlobalState::getInstance().m_withRegTerm, pow(GlobalState::getInstance().m_weightReg, 2),
												 GlobalState::getInstance().m_withRotTerm, pow(GlobalState::getInstance().m_weightRot, 2));
	m_gnSolver = new GNSolver(m_inputData, param);
	m_gnSolver->initCons(param);

	m_keyPoseVec.resize(MAX_FRAG_NUM);
	m_keyPoseInvVec.resize(MAX_FRAG_NUM);
	m_inputData->m_keyPoses = m_keyPoseVec.data();
	m_inputData->m_dKeyPoses = m_dKeyPoses;
	m_inputData->m_dUpdatedKeyPoses = m_dUpdatedKeyPoses;
	m_inputData->m_dUpdatedKeyPosesInv = m_dUpdatedKeyPosesInv;
	m_inputData->m_dKeyGrayImgs = m_dKeyGrayImgs.first;
	m_inputData->m_dKeyGrayImgsDx = m_dKeyGrayImgsDx.first;
	m_inputData->m_dKeyGrayImgsDy = m_dKeyGrayImgsDy.first;
}

void xDeformation::allocEssentialCUDAMemory()
{
	long long byteUsed = 0;

	m_dKeyGrayImgs.second = m_width * m_height;
	checkCudaErrors(hipMalloc(&m_dKeyGrayImgs.first, sizeof(float) * m_dKeyGrayImgs.second * MAX_FRAG_NUM));
	byteUsed += sizeof(float) * m_dKeyGrayImgs.second * MAX_FRAG_NUM;

	m_dKeyColorImgs.second = m_width * m_height * 3;
	checkCudaErrors(hipMalloc(&m_dKeyColorImgs.first, sizeof(uchar) * m_dKeyColorImgs.second * MAX_FRAG_NUM));
	byteUsed += sizeof(uchar) * m_dKeyColorImgs.second * MAX_FRAG_NUM;

	m_dKeyGrayImgsDx.second = m_width * m_height;
	checkCudaErrors(hipMalloc(&m_dKeyGrayImgsDx.first, sizeof(float) * m_dKeyGrayImgsDx.second * MAX_FRAG_NUM));
	byteUsed += sizeof(uchar) * m_dKeyGrayImgsDx.second * MAX_FRAG_NUM;

	m_dKeyGrayImgsDy.second = m_width * m_height;
	checkCudaErrors(hipMalloc(&m_dKeyGrayImgsDy.first, sizeof(float) * m_dKeyGrayImgsDy.second * MAX_FRAG_NUM));
	byteUsed += sizeof(uchar) * m_dKeyGrayImgsDy.second * MAX_FRAG_NUM;

	m_dIdxMaps.second = m_width * m_height;
	checkCudaErrors(hipMalloc(&m_dIdxMaps.first, sizeof(int) * m_dIdxMaps.second * MAX_FRAG_NUM));
	byteUsed += sizeof(int) * m_dIdxMaps.second * MAX_FRAG_NUM;

	checkCudaErrors(hipMalloc(&m_dIdxMapZBufs.first, m_dIdxMaps.second * sizeof(float) * MAX_FRAG_NUM));
	byteUsed += m_dIdxMaps.second * sizeof(float) * MAX_FRAG_NUM;

	checkCudaErrors(hipMalloc(&m_dMatchingPointIndices, sizeof(int) * 2 * MAX_CLOSURE_NUM_EACH_FRAG *
		SAMPLED_VERTEX_NUM_EACH_FRAG * MAX_FRAG_NUM));
	byteUsed += sizeof(int) * 2 * MAX_CLOSURE_NUM_EACH_FRAG * SAMPLED_VERTEX_NUM_EACH_FRAG * MAX_FRAG_NUM;

	checkCudaErrors(hipMalloc(&m_dMatchingFragIndices, sizeof(int) * 2 * MAX_CLOSURE_NUM_EACH_FRAG * MAX_FRAG_NUM));
	byteUsed += sizeof(int) * 2 * MAX_CLOSURE_NUM_EACH_FRAG * MAX_FRAG_NUM;

	checkCudaErrors(hipMalloc(&m_dKeyPoses, sizeof(float4) * 4 * MAX_FRAG_NUM));
	byteUsed += sizeof(float4) * 4 * MAX_FRAG_NUM;

	checkCudaErrors(hipMalloc(&m_dUpdatedKeyPosesInv, sizeof(float4) * 4 * MAX_FRAG_NUM));
	byteUsed += sizeof(float4) * 4 * MAX_FRAG_NUM;

	checkCudaErrors(hipMalloc(&m_dUpdatedKeyPoses, sizeof(float4) * 4 * MAX_FRAG_NUM));
	byteUsed += sizeof(float4) * 4 * MAX_FRAG_NUM;

	std::cout << "preallocated device memory used: " << byteUsed << std::endl;
}

// Using it when do not know the key frame
void xDeformation::addData(const cv::Mat& colorImg,
                           const cv::Mat& fullColorImg,
                           const cv::Mat_<uchar>& grayImg,
                           const cv::cuda::GpuMat& dGrayImg,
                           xMatrix4f pose)
{
	m_grayImgVec.push_back(grayImg.clone());
	m_colorImgVec.push_back(colorImg.clone());
	m_fullColorImgVec.push_back(fullColorImg.clone());

	//innoreal::InnoRealTimer timer;
	//timer.TimeStart();
#if 1
	//m_grayImgDevice.upload(grayImg);
	//float blurScore = CalculateBlurScoreGPU(m_grayImgDevice, m_horizontalBlurImgDevice, m_verticalBlurImgDevice);
	float blurScore = CalculateBlurScoreGPU(dGrayImg, m_dHorizontalBlurImg, m_dVerticalBlurImg);
#else
	float blurScore = CalculateBlurScore(grayImg);
#endif
	//timer.TimeEnd();
	//std::cout << "time blur: " << timer.TimeGap_in_ms() << std::endl;
	//std::cout << "blur score: " << blurScore << std::endl;
	m_blurScoreVec.push_back(blurScore);
	m_poseVec.push_back(pose);

	//std::cout << "blur: " << blurScore << std::endl;
	//std::cout << "pose: " << pose << std::endl;
}

// Using it when know the key frame, no need to calculate the blur score
void xDeformation::addDataWithKeyFrame(const cv::Mat& colorImg,
                                       const cv::Mat& rawDepthImg,
                                       const cv::Mat& grayImg,
                                       xMatrix4f pose)
{
	m_grayImgVec.push_back(grayImg.clone());
    m_depthImgVec.push_back(rawDepthImg.clone());
	m_colorImgVec.push_back(colorImg.clone());
	m_poseVec.push_back(pose);
}

void xDeformation::addDataWithKeyFrame(const cv::cuda::GpuMat& dGrayImg,
	xMatrix4f pose)
{
	m_dGrayImgVec.push_back(dGrayImg.clone());
	m_poseVec.push_back(pose);
}

void xDeformation::prepareData(int vertexNum)
{
	std::cout << "=======================" << std::endl;
	std::cout << "vertexNum: " << vertexNum << std::endl;
	std::cout << "=======================" << std::endl;

	int minBlurIdx = 0;
	float minBlurScore = 65536.0f;
	for (int i = 0; i < m_blurScoreVec.size(); ++i)
	{
		if (m_blurScoreVec[i] < minBlurScore)
		{
			minBlurScore = m_blurScoreVec[i];
			minBlurIdx = i;
		}
	}
    //std::cout << "1" << std::endl;

    m_keyFullColorImgVec.push_back(m_fullColorImgVec[minBlurIdx].clone()); 
    //std::cout << "2" << std::endl;

	cv::Mat keyGrayImgFloat;
	m_grayImgVec[minBlurIdx].convertTo(keyGrayImgFloat, CV_32FC1);
	cv::GaussianBlur(keyGrayImgFloat, keyGrayImgFloat, cv::Size(9, 9), 5, 5);
    //std::cout << "3" << std::endl;

	// prepare date for the current fragment
	checkCudaErrors(hipMemcpy(
		m_dKeyGrayImgs.first + m_fragIdx * m_dKeyGrayImgs.second,
		keyGrayImgFloat.data, m_dKeyGrayImgs.second * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(
		m_dKeyColorImgs.first + m_fragIdx * m_dKeyColorImgs.second,
		m_colorImgVec[minBlurIdx].data, m_dKeyColorImgs.second * sizeof(uchar), hipMemcpyHostToDevice));
    //std::cout << "4" << std::endl;

	cv::Mat_<float> dxMat, dyMat;
	cv::sepFilter2D(keyGrayImgFloat, dxMat, dxMat.depth(), m_kxRow, m_kyRow);
	cv::sepFilter2D(keyGrayImgFloat, dyMat, dyMat.depth(), m_kxCol, m_kyCol);

	checkCudaErrors(hipMemcpy(
		m_dKeyGrayImgsDx.first + m_fragIdx * m_dKeyGrayImgsDx.second,
		dxMat.data, m_width * m_height * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(
		m_dKeyGrayImgsDy.first + m_fragIdx * m_dKeyGrayImgsDy.second,
		dyMat.data, m_width * m_height * sizeof(float), hipMemcpyHostToDevice));

	m_keyPoseVec[m_fragIdx] = m_poseVec[minBlurIdx];
	m_keyPoseInvVec[m_fragIdx] = m_poseVec[minBlurIdx].inverse();
	//m_keyPoseVec.push_back(m_poseVec[minBlurInd]);
	//m_keyPoseInvVec.push_back(m_poseVec[minBlurInd].inverse());
	checkCudaErrors(hipMemcpy(
		m_dKeyPoses + m_fragIdx * 4,
		m_keyPoseVec[m_fragIdx].data(), sizeof(float4) * 4, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(
		m_dUpdatedKeyPoses + m_fragIdx * 4,
		m_keyPoseVec[m_fragIdx].data(), sizeof(float4) * 4, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(
		m_dUpdatedKeyPosesInv + m_fragIdx * 4,
		m_keyPoseInvVec[m_fragIdx].data(), sizeof(float4) * 4, hipMemcpyHostToDevice));

#if 0
	int vertexNumFrag = m_vertexStrideVec[m_fragInd + 1] - m_vertexStrideVec[m_fragInd];
	VertexFilter(m_vboDevice + m_vertexStrideVec[m_fragInd], vertexNumFrag, m_keyPosesDevice + m_fragInd * 4,
		Intrinsics::getInstance().fx(), Intrinsics::getInstance().fy(), Intrinsics::getInstance().cx(), Intrinsics::getInstance().cy());
#endif

#if 0
	std::cout << "minBlurScore: " << minBlurScore << std::endl;
	if (minBlurScore < 0.315)
	{
		m_isFragValid[fragInd] = false;
		goto invalid_fragment;
	}
#endif

	m_poseGraph.push_back(std::vector<int>());
	m_poseGraphInv.push_back(std::vector<int>());
	updatePoseGraph();

	m_inputData->prepareData(m_dVboCuda, vertexNum, m_fragIdx);
	m_gnSolver->initVars();
}

void xDeformation::prepareDataWithKeyFrame(int vertexNum, int keyFrameIdxEachFrag)
{
	//std::cout << "=======================" << std::endl;
	//std::cout << "vertexNum: " << vertexNum << std::endl;
	//std::cout << "=======================" << std::endl;

	//cv::Mat grayImg;
	//char fileDir[256];
	//m_dGrayImgVec[keyFrameIdxEachFrag].download(grayImg);
	//sprintf(fileDir, "C:\\xjm\\snapshot\\before_opt\\%06d_test.png", m_fragIdx);
	//cv::imwrite(fileDir, grayImg);

	if (GlobalState::getInstance().m_withPhotoTerm) {
		m_dGrayImgVec[keyFrameIdxEachFrag].convertTo(m_dKeyGrayImgFloat, CV_32FC1);
		m_gaussFilter->apply(m_dKeyGrayImgFloat, m_dKeyGrayImgFloat);
		m_dKeyGrayImgFloat.download(m_keyGrayImgFloat);
		//m_keyDepthImgVec.push_back(m_depthImgVec[keyFrameIdxEachFrag]);

	// prepare date for the current fragment
		checkCudaErrors(hipMemcpy(
			m_dKeyGrayImgs.first + m_fragIdx * m_dKeyGrayImgs.second,
			m_keyGrayImgFloat.data, m_dKeyGrayImgs.second * sizeof(float), hipMemcpyHostToDevice));

		cv::Mat_<float> dxMat, dyMat;
		cv::sepFilter2D(m_keyGrayImgFloat, dxMat, dxMat.depth(), m_kxRow, m_kyRow);
		cv::sepFilter2D(m_keyGrayImgFloat, dyMat, dyMat.depth(), m_kxCol, m_kyCol);

		checkCudaErrors(hipMemcpy(
			m_dKeyGrayImgsDx.first + m_fragIdx * m_dKeyGrayImgsDx.second,
			dxMat.data, m_width * m_height * sizeof(float), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(
			m_dKeyGrayImgsDy.first + m_fragIdx * m_dKeyGrayImgsDy.second,
			dyMat.data, m_width * m_height * sizeof(float), hipMemcpyHostToDevice));
	}

	m_keyPoseVec[m_fragIdx] = m_poseVec[keyFrameIdxEachFrag];
	m_keyPoseInvVec[m_fragIdx] = m_poseVec[keyFrameIdxEachFrag].inverse();
	//m_keyPoseVec.push_back(m_poseVec[minBlurInd]);
	//m_keyPoseInvVec.push_back(m_poseVec[minBlurInd].inverse());
	checkCudaErrors(hipMemcpy(
		m_dKeyPoses + m_fragIdx * 4,
		m_keyPoseVec[m_fragIdx].data(), sizeof(float4) * 4, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(
		m_dUpdatedKeyPoses + m_fragIdx * 4,
		m_keyPoseVec[m_fragIdx].data(), sizeof(float4) * 4, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(
		m_dUpdatedKeyPosesInv + m_fragIdx * 4,
		m_keyPoseInvVec[m_fragIdx].data(), sizeof(float4) * 4, hipMemcpyHostToDevice));

	m_poseGraph.push_back(std::vector<int>());
	m_poseGraphInv.push_back(std::vector<int>());
	updatePoseGraph();

	m_inputData->prepareData(m_dVboCuda, vertexNum, m_fragIdx);
	m_gnSolver->initVars();
}

void xDeformation::deform(xMatrix4f* latestPose, VBOType* vboDevice, int vertexNum, int keyFrameIdxEachFrag)
{
	//innoreal::InnoRealTimer timer;
#if USE_STRUCTURE_SENSOR 
	//timer.TimeStart();
	prepareDataWithKeyFrame(vertexNum, keyFrameIdxEachFrag);
	//timer.TimeEnd();
	//printf("prepare data time: %f\n", timer.TimeGap_in_ms());
#endif
#if USE_XTION
	prepareData(vertexNum);
#endif

	// do optimization
	//std::cout << "fragInd: " << m_fragIdx << std::endl;
	//std::cout << "loopClosureNum: " << m_loopClosureNum << std::endl;
	int iterNum = 3;
	xMatrix4f keyPose;
	xMatrix4f updatedKeyPose;
	if (m_loopClosureNum > 0) // && m_fragInd > 1)
	{
		//innoreal::InnoRealTimer timer;
		//timer.TimeStart();
		//std::cout << "do optimize" << std::endl;
		if (GlobalState::getInstance().m_doNonrigidRegistration) 
		{
			doOptimize(iterNum);
		}
		else
		{
			//doOptimize(1);
		}
		//std::cout << "do optimize finish" << std::endl;
		//timer.TimeEnd();
		//printf("do optimize time: %f\n", timer.TimeGap_in_ms());

		checkCudaErrors(hipMemcpy(
			updatedKeyPose.data(),
			m_dUpdatedKeyPoses + m_fragIdx * 4, sizeof(float4) * 4, hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(
			keyPose.data(),
			m_dKeyPoses + m_fragIdx * 4, sizeof(float4) * 4, hipMemcpyDeviceToHost));
		*latestPose = updatedKeyPose * keyPose.inverse() * (*latestPose);
		//keyPose.print();
		//updatedKeyPose.print();
		//latestPose->print();
	}

invalid_fragment:
	m_grayImgVec.clear();
	m_dGrayImgVec.clear();
	m_depthImgVec.clear();
	m_colorImgVec.clear();
	m_fullColorImgVec.clear();
	m_blurScoreVec.clear();
	m_poseVec.clear();
}

static bool AngleVecCompare(const std::pair<float, int>& a, const std::pair<float, int>& b)
{
	return a.first > b.first;
}

void xDeformation::updatePoseGraph()
{
	int latestPoseIdx = m_fragIdx;
	//int latestPoseIdx = m_keyPoseVec.size() - 1;
	xMatrix4f& latestPoseMat = m_keyPoseVec[latestPoseIdx];
	float4 latestCamOrient = m_keyPoseVec[latestPoseIdx].col(2), camOrient;
	xMatrix4f ralaTrans;

	std::vector<std::pair<float, int>> angleVec;

	for (int i = latestPoseIdx - 1; i >= 0; --i)
	{
		camOrient = m_keyPoseVec[i].col(2);
		if (m_isFragValid[i] > 0 && dot(camOrient, latestCamOrient) > -0.5)
		{
			angleVec.push_back(std::make_pair(dot(camOrient, latestCamOrient), i));
		}
	}
	std::sort(angleVec.begin(), angleVec.end(), AngleVecCompare);
#if 0
	for (int i = 0; i < angleVec.size(); ++i)
	{
		std::cout << angleVec[i].second << std::endl;
	}
#endif
	for (int i = 0; i < MIN(angleVec.size(), MAX_CLOSURE_NUM_EACH_FRAG); ++i)
	{
		m_poseGraph[angleVec[i].second].push_back(latestPoseIdx);
		m_poseGraphInv[latestPoseIdx].push_back(angleVec[i].second);
		++m_loopClosureNum;
	}
	//std::cout << "loop closure num: " << m_loopClosureNum << std::endl;
}

void xDeformation::findMatchingKNN()
{
	FragDeformableMeshData& sourceMesh = m_inputData->m_source;
	MeshData& deformedMesh = m_inputData->m_deformed;

	int* sampledVertexIndicesDeviceSrcFrag;
	int* matchingPointsFrag;
	int vertexNumSrc, sampledVertexNumSrc, vertexNumTarget, vertexIndBaseTarget;

	float4* sampledUpdatedVertexPosesSrc;
	checkCudaErrors(hipMalloc(&sampledUpdatedVertexPosesSrc, sizeof(float4) * SAMPLED_VERTEX_NUM_EACH_FRAG));
	int* knnIndex;
	float* knnWeight;
	checkCudaErrors(hipMalloc(&knnIndex, sizeof(int) * SAMPLED_VERTEX_NUM_EACH_FRAG));
	checkCudaErrors(hipMalloc(&knnWeight, sizeof(float) * SAMPLED_VERTEX_NUM_EACH_FRAG));

	int srcFragInd;
	int matchingVertexNum;
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipGetLastError());
	m_matchingPointNum = m_matchingPointsNumDescriptor;
	for (int targetFragInd = 0; targetFragInd < m_poseGraph.size(); ++targetFragInd)
	{
		vertexNumTarget = sourceMesh.m_vertexStrideVec[targetFragInd + 1] - sourceMesh.m_vertexStrideVec
			[targetFragInd];
		vertexIndBaseTarget = sourceMesh.m_vertexStrideVec[targetFragInd];

		NearestPoint nearestPoint;
		nearestPoint.InitKDTree(
			RAW_PTR(deformedMesh.m_dVertexVec) + sourceMesh.m_vertexStrideVec[targetFragInd],
			vertexNumTarget);
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipGetLastError());

		for (int ind = 0; ind < m_poseGraph[targetFragInd].size(); ++ind)
		{
			srcFragInd = m_poseGraph[targetFragInd][ind];
			if (m_isFragValid[srcFragInd] <= 0 || m_isFragValid[targetFragInd] <= 0)
			{
				continue;
			}
#if 0
			std::cout << "src: " << srcFragInd << std::endl;
			std::cout << "target: " << targetFragInd << std::endl;
			std::cout << "vertexIdBaseTarget: " << vertexIndBaseTarget << std::endl;
#endif
			sampledVertexNumSrc = SAMPLED_VERTEX_NUM_EACH_FRAG;
			matchingPointsFrag = m_dMatchingPointIndices + 2 * m_matchingPointNum;
			m_matchingPointNum += sampledVertexNumSrc;

			CompressSampledVertex(sampledUpdatedVertexPosesSrc, RAW_PTR(deformedMesh.m_dVertexVec),
			                      RAW_PTR(sourceMesh.m_dSampledVertexIdxVec) + SAMPLED_VERTEX_NUM_EACH_FRAG *
			                      srcFragInd, sampledVertexNumSrc);
			checkCudaErrors(hipDeviceSynchronize());
			checkCudaErrors(hipGetLastError());

			nearestPoint.GetKnnResult(sampledUpdatedVertexPosesSrc, sampledVertexNumSrc, 1,
			                          knnIndex,
			                          knnWeight);
			checkCudaErrors(hipDeviceSynchronize());
			checkCudaErrors(hipGetLastError());

			AddToMatchingPoints(matchingPointsFrag,
			                    RAW_PTR(sourceMesh.m_dSampledVertexIdxVec) + SAMPLED_VERTEX_NUM_EACH_FRAG *
			                    srcFragInd,
			                    knnIndex, vertexIndBaseTarget, sampledVertexNumSrc);
		}
	}
	m_matchingPointsNumNearest = m_matchingPointNum - m_matchingPointsNumDescriptor;
	checkCudaErrors(hipFree(sampledUpdatedVertexPosesSrc));
	checkCudaErrors(hipFree(knnIndex));
	checkCudaErrors(hipFree(knnWeight));
}

void xDeformation::findMatchingPerspective()
{
	FragDeformableMeshData& sourceMesh = m_inputData->m_source;
	MeshData& deformedMesh = m_inputData->m_deformed;

	int vertexNum = sourceMesh.m_vertexStrideVec[m_fragIdx + 1];
	m_matchingPointNum = m_matchingPointsNumDescriptor;

	std::vector<int> matchingFragsIndicesVec;
	matchingFragsIndicesVec.reserve(100);
	int srcFragInd;
	for (int targetFragInd = 0; targetFragInd < m_poseGraph.size(); ++targetFragInd)
	{
		for (int ind = 0; ind < m_poseGraph[targetFragInd].size(); ++ind)
		{
			srcFragInd = m_poseGraph[targetFragInd][ind];
			//std::cout << srcFragInd << " : " << targetFragInd << std::endl;
			matchingFragsIndicesVec.push_back(srcFragInd);
			matchingFragsIndicesVec.push_back(targetFragInd);
		}
	}
	assert(m_loopClosureNum == (matchingFragsIndicesVec.size() / 2));
	checkCudaErrors(hipMemcpy(m_dMatchingFragIndices, matchingFragsIndicesVec.data(),
		matchingFragsIndicesVec.size() * sizeof(int), hipMemcpyHostToDevice));

	UpdateIndMapsPerspective(m_dIdxMaps.first,
	                         m_dIdxMapZBufs.first,
	                         m_width, m_height, m_fragIdx + 1, vertexNum,
	                         Intrinsics::getInstance().fx(), Intrinsics::getInstance().fy(),
	                         Intrinsics::getInstance().cx(), Intrinsics::getInstance().cy(),
	                         RAW_PTR(deformedMesh.m_dVertexVec),
	                         RAW_PTR(deformedMesh.m_dNormalVec),
	                         m_dUpdatedKeyPosesInv);

	m_matchingPointsNumNearest = m_loopClosureNum * SAMPLED_VERTEX_NUM_EACH_FRAG;
	FindMatchingPointsPerspective(
		m_dMatchingPointIndices + 2 * m_matchingPointNum,
		m_dMatchingFragIndices,
		RAW_PTR(deformedMesh.m_dVertexVec),
		m_dIdxMaps,
		m_dUpdatedKeyPosesInv,
		RAW_PTR(sourceMesh.m_dSampledVertexIdxVec),
		m_width, m_height, Intrinsics::getInstance().fx(), Intrinsics::getInstance().fy(), Intrinsics::getInstance().cx(),
		Intrinsics::getInstance().cy(),
		m_matchingPointsNumNearest);

	m_matchingPointNum += m_matchingPointsNumNearest;
}

void xDeformation::doOptimize(int iterNum)
{
	//innoreal::InnoRealTimer timer;
	for (int iter = 0; iter < iterNum; ++iter)
	{
		//timer.TimeStart();
		findMatchingPoints();	
		FilterInvalidMatchingPoints(
			m_dMatchingPointIndices,
			m_matchingPointsNumDescriptor,
			m_matchingPointNum,
			RAW_PTR(m_inputData->m_deformed.m_dVertexVec),
			RAW_PTR(m_inputData->m_deformed.m_dNormalVec),
			iter);
		m_inputData->m_dMatchingPointIndices = m_dMatchingPointIndices;
		m_inputData->m_matchingPointNum = m_matchingPointNum;
		//timer.TimeEnd();
		//printf("fine matching time: %f\n", timer.TimeGap_in_ms());
		//timer.TimeStart();
		m_inputData->getIijSet(m_dMatchingPointIndices, m_matchingPointNum);
		//timer.TimeEnd();
		//printf("get Iij time: %f\n", timer.TimeGap_in_ms());

		//timer.TimeStart();
		m_gnSolver->initJtj();
		m_gnSolver->next(iter);
		//timer.TimeEnd();
		//printf("solve time: %f\n", timer.TimeGap_in_ms());
	}
	m_gnSolver->updateVboVec(m_dVboCuda);

	return;
}

void xDeformation::savePly(const char* fileDir, int fragIdx)
{
	FragDeformableMeshData& sourceMesh = m_inputData->m_source;
	MeshData& deformedMesh = m_inputData->m_deformed;
	int vertexNum;
	if (fragIdx == -1)
	{
		vertexNum = sourceMesh.m_vertexStrideVec[sourceMesh.m_fragNum];
	}
	else
	{
		vertexNum = sourceMesh.m_vertexStrideVec[fragIdx + 1] - sourceMesh.m_vertexStrideVec[fragIdx];
	}
	std::cout << "save ply\nvertexNum: " << vertexNum << std::endl;
	std::vector<VBOType> vboCudaVec(vertexNum);

	std::ofstream fs;
	fs.open(fileDir);
	if (fragIdx == -1)
	{
		checkCudaErrors(hipMemcpy(vboCudaVec.data(),
			m_dVboCuda,
			vboCudaVec.size() * sizeof(VBOType), hipMemcpyDeviceToHost));
	}
	else
	{
		checkCudaErrors(hipMemcpy(vboCudaVec.data(),
			m_dVboCuda + sourceMesh.m_vertexStrideVec[fragIdx],
			vboCudaVec.size() * sizeof(VBOType), hipMemcpyDeviceToHost));
	}

	int validVertexNumFrag = 0;
	for (unsigned int i = 0; i < vertexNum; i++)
	{
		VBOType& posColorNor = vboCudaVec[i];
		if (posColorNor.colorTime.y >= 0)
		{
			++validVertexNumFrag;
		}
	}

	fs << "ply";
	fs << "\nformat " << "ascii" << " 1.0";
	fs << "\nelement vertex " << validVertexNumFrag;
	fs << "\nproperty float x"
		"\nproperty float y"
		"\nproperty float z";
	fs << "\nproperty uchar red"
		"\nproperty uchar green"
		"\nproperty uchar blue";
	fs << "\nproperty float nx"
		"\nproperty float ny"
		"\nproperty float nz";
	fs << "\nend_header\n";

	int cnt = 0;
	int randNum_0 = rand() % 3;
	int randNum_1 = rand() % 250 - 50;
	for (unsigned int i = 0; i < validVertexNumFrag; i++)
	{
		VBOType& posColorNor = vboCudaVec[i];

		if (posColorNor.colorTime.y >= 0)
		{
			unsigned char b, g, r;
			b = int(posColorNor.colorTime.x) >> 16 & 0xFF;
			g = int(posColorNor.colorTime.x) >> 8 & 0xFF;
			r = int(posColorNor.colorTime.x) & 0xFF;
			fs << posColorNor.posConf.x << " " << posColorNor.posConf.y << " " << posColorNor.posConf.z << " "
				<< (int)r << " " << (int)g << " " << (int)b << " "
				<< -posColorNor.normalRad.x << " " << -posColorNor.normalRad.y << " " << -posColorNor.normalRad.z
				<< std::endl;
		}
	}

	fs.close();
}

void xDeformation::saveModel(int m_fragNumInDetectStage)
{
	std::cout << "save model" << std::endl;

	FragDeformableMeshData& sourceMesh = m_inputData->m_source;
	MeshData& deformedMesh = m_inputData->m_deformed;

	char fileDir[256];
	sprintf(fileDir, "C:\\xjm\\snapshot\\before_opt\\whole_model.ply");
	savePly(fileDir, -1);
#if 1
	for (int i = 0; i < sourceMesh.m_fragNum; ++i)
	{
		sprintf(fileDir, "C:\\xjm\\snapshot\\before_opt\\%06d.ply", i);
		savePly(fileDir, i);
	}
#endif

	int width = Resolution::getInstance().width();
	int height = Resolution::getInstance().height();
    cv::Mat keyColorImg(height, width, CV_8UC3);
    //cv::Mat keyGrayImg(height, width, CV_8UC1);
	//cv::Mat keyColorImgResized(height, width, CV_8UC3);
	std::vector<int> pngCompressionParams;
	pngCompressionParams.push_back(CV_IMWRITE_PNG_COMPRESSION);
	pngCompressionParams.push_back(0);
	std::ofstream fs1, fs2;
	fs1.open("C:\\xjm\\snapshot\\before_opt\\camera_pose.txt", std::ofstream::binary);
	fs2.open("C:\\xjm\\snapshot\\before_opt\\camera_pose_original.txt", std::ofstream::binary);

	float4 camPose[4], oriCamPose[4], invCamPose[4];
	for (int fragIdx = 0; fragIdx < sourceMesh.m_fragNum; ++fragIdx)
	{
		checkCudaErrors(hipMemcpy(camPose,
										m_inputData->m_dUpdatedKeyPoses + 4 * fragIdx,
										4 * sizeof(float4), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(oriCamPose,
										m_inputData->m_dKeyPoses + 4 * fragIdx,
										4 * sizeof(float4), hipMemcpyDeviceToHost));
#if 0
		checkCudaErrors(hipMemcpy(invCamPose,
			m_inputData->m_dUpdatedKeyPosesInv + 4 * fragIdx,
			4 * sizeof(float4), hipMemcpyDeviceToHost));
#endif
		std::cout << "fragIdx: " << fragIdx << std::endl;
		std::cout << "ori camera pose:\n" <<
			oriCamPose[0].x << " " << oriCamPose[0].y << " " << oriCamPose[0].z << " " << oriCamPose[0].w <<
			oriCamPose[1].x << " " << oriCamPose[1].y << " " << oriCamPose[1].z << " " << oriCamPose[1].w <<
			oriCamPose[2].x << " " << oriCamPose[2].y << " " << oriCamPose[2].z << " " << oriCamPose[2].w <<
			oriCamPose[3].x << " " << oriCamPose[3].y << " " << oriCamPose[3].z << " " << oriCamPose[3].w << std::endl;
		std::cout << "camera pose:\n" <<
			camPose[0].x << " " << camPose[0].y << " " << camPose[0].z << " " << camPose[0].w <<
			camPose[1].x << " " << camPose[1].y << " " << camPose[1].z << " " << camPose[1].w <<
			camPose[2].x << " " << camPose[2].y << " " << camPose[2].z << " " << camPose[2].w <<
			camPose[3].x << " " << camPose[3].y << " " << camPose[3].z << " " << camPose[3].w << std::endl;
		int fragIdxWithDetectStage = m_fragNumInDetectStage + fragIdx;
		fs1.write((char *)&fragIdxWithDetectStage, sizeof(int));
		fs1.write((char *)camPose, 4 * sizeof(float4));
		fs2.write((char *)&fragIdxWithDetectStage, sizeof(int));
		fs2.write((char *)oriCamPose, 4 * sizeof(float4));

#if 0
        sprintf(fileDir, "C:\\xjm\\snapshot\\before_opt\\%06d_key_depth.png", fragIdx);
        cv::imwrite(fileDir, m_keyDepthImgVec[fragIdx], pngCompressionParams);
#endif
#if 0
        checkCudaErrors(hipMemcpy(keyColorImg.data,
            m_dKeyColorImgs.first + fragIdx * m_dKeyColorImgs.second,
            keyColorImg.rows * keyColorImg.cols * 3,
            hipMemcpyDeviceToHost));
        sprintf(fileDir, "C:\\xjm\\snapshot\\before_opt\\%06d_key_frame.png", fragIdx);
        cv::imwrite(fileDir, keyColorImg, pngCompressionParams);
#endif
       
#if 0
        sprintf(fileDir, "D:\\xjm\\result\\before_opt\\%06d_key_frame.png", fragIdx);
        cv::imwrite(fileDir, m_keyFullColorImgVec[fragIdx], pngCompressionParams);
#endif

		std::cout << "Frag " << fragIdx << " has been saved" << std::endl;
	}
	fs1.close();
	fs2.close();
	//exit(0);
}

int xDeformation::getSrcVertexNum()
{
	return m_inputData->getSrcVertexNum();
}

int xDeformation::getSrcNodeNum()
{
	return m_inputData->getSrcNodeNum();
}

int xDeformation::getFragNum()
{
	return m_fragIdx + 1;
}

void xDeformation::getVertexStrideVe(std::vector<int>& vertexStrideVec)
{
	vertexStrideVec = m_inputData->m_source.m_vertexStrideVec;
}

void xDeformation::getDeformedVertices(std::vector<float4>& deformedVertexVec)
{
	deformedVertexVec.resize(m_inputData->m_deformed.m_vertexNum);

	checkCudaErrors(hipMemcpy(deformedVertexVec.data(), RAW_PTR(m_inputData->m_deformed.m_dVertexVec),
		deformedVertexVec.size() * sizeof(float4), hipMemcpyDeviceToHost));
}

void xDeformation::getDeformedNormals(std::vector<float4>& deformedVertexVec)
{
	deformedVertexVec.resize(this->m_inputData->m_deformed.m_vertexNum);

	checkCudaErrors(hipMemcpy(deformedVertexVec.data(), RAW_PTR(m_inputData->m_deformed.m_dNormalVec),
		deformedVertexVec.size() * sizeof(float4), hipMemcpyDeviceToHost));
}

void xDeformation::getMatchingPointIndices(std::vector<int>& matchingPointIdxVec)
{
	matchingPointIdxVec.resize(m_inputData->m_matchingPointNum * 2);

	checkCudaErrors(hipMemcpy(matchingPointIdxVec.data(), RAW_PTR(m_inputData->m_dMatchingPointIndices),
		matchingPointIdxVec.size() * sizeof(int), hipMemcpyDeviceToHost));
}

